// Copyright (c) 2019, Zpalmtree
//
// Please see the included LICENSE file for more information.

#include <iostream>
#include <map>
#include <vector>

#include "Nvidia/Argon2.h"
#include "Utilities/ColouredMsg.h"

int getDeviceCount()
{
    int numberDevices;
    hipGetDeviceCount(&numberDevices);
    return numberDevices;
}

std::string getDeviceName(uint16_t deviceId)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceId);
    return prop.name;
}

std::vector<std::tuple<std::string, bool, int>> getNvidiaDevicesActual()
{
    std::vector<std::tuple<std::string, bool, int>> devices;

    int numberDevices = getDeviceCount();

    for (int i = 0; i < numberDevices; i++)
    {
        devices.push_back(std::make_tuple(getDeviceName(i), true, i));
    }

    return devices;
}

void printNvidiaHeader()
{
    std::cout << InformationMsg<std::string>("* ") << WhiteMsg<std::string>("NVIDIA DEVICES", 25);

    int numberDevices = getDeviceCount();

    if (numberDevices == 0)
    {
        std::cout << WarningMsg<std::string>("None found") << std::endl;
        return;
    }

    std::map<std::string, std::vector<hipDeviceProp_t>> gpus;

    for (int i = 0; i < numberDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        std::string deviceName = prop.name;

        gpus[deviceName].push_back(prop);
    }

    int i = 0;

    for (const auto gpu : gpus)
    {
        const auto name = gpu.first;
        const auto properties = gpu.second;

        if (i != 0)
        {
            std::cout << ", ";
        }

        size_t numDevices = properties.size();

        std::cout << SuccessMsg<size_t>(numDevices) << SuccessMsg<std::string>("x ") << SuccessMsg<std::string>(name);

        i++;
    }

    std::cout << std::endl;
}
